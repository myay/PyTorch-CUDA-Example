#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__global__ void test_cpp_cuda_kernel(
    torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> input) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
    input[c] += 1;
}

std::vector<torch::Tensor> test_cpp_cuda(torch::Tensor input) {
  const int input_size = input.size(0);
  const int threads = 256;
  const int batch_size = 1;
  const dim3 blocks((input_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "test_cpp_cuda", ([&] {
    test_cpp_cuda_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>());
  }));

  return {input};
}
